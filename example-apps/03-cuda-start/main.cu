// Main file had to be main.cu and not main.cpp otherwise <<< is not recognized.
// I was able to overcome this in VS Code by compiling CU files via NVCC into a library and compile main.cpp via MSVC and link them later
#include "vector_add.h"

#include <Workshop/Shader.hpp>
#include <Workshop/Workshop.hpp>

#include <glad/gl.h>
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include <imgui.h>
#include <glm/gtc/type_ptr.hpp>
#include <glm/vec3.hpp>

// #include <implot.h>
// #include <stb_image.h>
// #include <tiny_obj_loader.h>
// #include <vivid/vivid.h>

#include <iostream>

int main(int argc, char* argv[]) {
  const int count = (argc == 2) ? std::stoi(argv[1]) : 5;
  const size_t size = count * sizeof(float);

  // Allocate memory of inputs on host, fill them out
  std::vector<float> h_a(count), h_b(count), h_c(count);
  for (int ix = 0; ix < count; ++ix) {
    h_a[ix] = ix;
    h_b[ix] = 2 * ix;
  }

  // Allocate memory for device counterparts
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Copy inputs from host to device
  hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice);

  // Run kernel
  vectorAdd<<<1, size>>>(d_a, d_b, d_c);

  // Copy the result back to host
  hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost);

  // "Use the result"
  for (int i = 0; i < count; ++i)
    std::cout << std::format("{} + {} = {}\n", h_a[i], h_b[i], h_c[i]);

  // Free memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "Hi!\n";
  ws::Workshop workshop{800, 600, "Workshop App"};

  const char *vertexShader = R"(
#version 300 es
#extension GL_EXT_separate_shader_objects : enable
precision mediump float;

layout (location = 0) out vec3 fragColor;

vec2 positions[3] = vec2[](vec2 (0.0, -0.5), vec2 (0.5, 0.5), vec2 (-0.5, 0.5));
vec3 colors[3] = vec3[](vec3 (1.0, 0.0, 0.0), vec3 (0.0, 1.0, 0.0), vec3 (0.0, 0.0, 1.0));
void main ()
{
	gl_Position = vec4 (positions[gl_VertexID], 0.0, 1.0);
	fragColor = colors[gl_VertexID];
}
  )";

  const char *fragmentShader = R"(
#version 300 es
#extension GL_EXT_separate_shader_objects : enable
precision mediump float;

layout (location = 0) in vec3 fragColor;
layout (location = 0) out vec4 outColor;

void main () { outColor = vec4 (fragColor, 1.0); }
  )";
  ws::Shader shader{vertexShader, fragmentShader};

  while (!workshop.shouldStop())
  {
    workshop.beginFrame();

    ImGui::Begin("Main");
    static bool shouldShowImGuiDemo = false;
    ImGui::Checkbox("Show Demo", &shouldShowImGuiDemo);
    if (shouldShowImGuiDemo)
      ImGui::ShowDemoWindow();

    static glm::vec3 bgColor{42 / 256.0, 96 / 256.0, 87 / 256.0};
    ImGui::ColorEdit3("BG Color", glm::value_ptr(bgColor));
    ImGui::End();

    // VAO binding is needed in 4.6 was not needed in 3.1
    uint32_t vao;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    glClearColor(bgColor.x, bgColor.y, bgColor.z, 1);
    glClear(GL_COLOR_BUFFER_BIT);

    shader.bind();
    glDrawArrays(GL_TRIANGLES, 0, 3);

    workshop.endFrame();
  }

  return 0;
}