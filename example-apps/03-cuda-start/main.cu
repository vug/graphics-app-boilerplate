// Main file had to be main.cu and not main.cpp otherwise <<< is not recognized.
// I was able to overcome this in VS Code by compiling CU files via NVCC into a library and compile main.cpp via MSVC and link them later
#include "kernels.h"

#include <Workshop/Shader.hpp>
#include <Workshop/Texture.hpp>
#include <Workshop/Workshop.hpp>

#include <glad/gl.h>
#define GLFW_INCLUDE_NONE
#include <GLFW/glfw3.h>
#include <imgui.h>
#include <glm/gtc/type_ptr.hpp>
#include <glm/vec3.hpp>

// #include <implot.h>
// #include <stb_image.h>
// #include <tiny_obj_loader.h>
// #include <vivid/vivid.h>

#include <iostream>

void calcPixelsCpuToTex(ws::Texture& tex, const glm::uvec2& ws) {

  std::vector<uint32_t> pixels(ws.x * ws.y);
  for (uint32_t i = 0; i < ws.y; ++i) {
    for (uint32_t j = 0; j < ws.x; ++j) {
      // Red: 0xFF0000FF, Green: 0xFF00FF00, Blue: 0xFFFF0000
      const uint8_t red = j * 255 / ws.x;
      const uint8_t green = i * 255 / ws.y;
      const uint8_t blue = 0;
      const uint8_t alpha = 255;
      pixels[i * ws.x + j] = (alpha << 24) + (blue << 16) + (green << 8) + red;
    }
  }
  tex.loadPixels(pixels.data());
}

void calcPixelsGpuToCpuToTex(ws::Texture& tex, const glm::uvec2& ws) {
  std::vector<uint32_t> pixels(ws.x * ws.y);
  size_t texSizeBytes = pixels.size() * sizeof(uint32_t);
  uint32_t* d_pixels;
  hipMalloc(&d_pixels, texSizeBytes);

  const auto threadSize = dim3(32, 32);
  const auto blockSize = dim3(ws.x / threadSize.x + 1, ws.y / threadSize.y + 1);
  printf("numPixels: %d, sizeBytes: %d, blockSize (%d, %d), threadSize: (%d, %d)\n", pixels.size(), texSizeBytes, blockSize.x, blockSize.y, threadSize.x, threadSize.y);
  genTexture<<<blockSize, threadSize>>>(d_pixels, ws.x, ws.y);
  hipDeviceSynchronize();
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    printf("CUDA error: %s\n", hipGetErrorString(error));
    exit(-1);
  }

  hipMemcpy(pixels.data(), d_pixels, texSizeBytes, hipMemcpyDeviceToHost);
  hipFree(d_pixels);

  tex.loadPixels(pixels.data());
}

void calcPixelsGlInterop(ws::Texture& tex, const glm::uvec2& ws) {

}

int main(int argc, char* argv[]) {
  const int count = (argc == 2) ? std::stoi(argv[1]) : 5;
  const size_t size = count * sizeof(float);

  // Allocate memory of inputs on host, fill them out
  std::vector<float> h_a(count), h_b(count), h_c(count);
  for (int ix = 0; ix < count; ++ix) {
    h_a[ix] = ix;
    h_b[ix] = 2 * ix;
  }

  // Allocate memory for device counterparts
  float *d_a, *d_b, *d_c;
  hipMalloc(&d_a, size);
  hipMalloc(&d_b, size);
  hipMalloc(&d_c, size);

  // Copy inputs from host to device
  hipMemcpy(d_a, h_a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b.data(), size, hipMemcpyHostToDevice);

  // Run kernel
  vectorAdd<<<1, size>>>(d_a, d_b, d_c);

  // Copy the result back to host
  hipMemcpy(h_c.data(), d_c, size, hipMemcpyDeviceToHost);

  // "Use the result"
  for (int i = 0; i < count; ++i)
    std::cout << std::format("{} + {} = {}\n", h_a[i], h_b[i], h_c[i]);

  // Free memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "Hi!\n";
  ws::Workshop workshop{800, 600, "Workshop App"};

  const char *vertexShader = R"(
#version 460

out VertexData {
  vec2 uv;
} v; // vertex-to-fragment or vertex-to-geometry

vec2 positions[4] = vec2[](vec2(-1, -1), vec2(1, -1), vec2(1, 1), vec2(-1, 1));
vec2 uvs[4] = vec2[](vec2(0, 0), vec2(1, 0), vec2(1, 1), vec2(0, 1));
int indices[6] = int[](0, 1, 2, 0, 2, 3);

void main () {
  int ix = indices[gl_VertexID];
	gl_Position = vec4 (positions[ix], 0.0, 1.0);
	v.uv = uvs[ix];
}
  )";

  const char *fragmentShader = R"(
#version 460

in VertexData {
  vec2 uv;
} v;

uniform sampler2D screenTexture;

layout (location = 0) out vec4 outColor;

void main () {
  //outColor = vec4(v.uv.x, v.uv.y, 0, 1.0); 

  vec3 tex =  texture(screenTexture, v.uv).rgb;
  outColor.rgb = tex;

  //float val = (tex.r + tex.g + tex.b) / 3.0;
  //outColor.rgb = vec3(val);
}
  )";
  ws::Shader shader{vertexShader, fragmentShader};
  const auto ws = workshop.getWindowSize();
  auto desc = ws::Texture::Specs {ws.x, ws.y, ws::Texture::Format::RGBA8};
  ws::Texture tex{desc};

  //calcPixelsCpuToTex(tex, ws);
  calcPixelsGpuToCpuToTex(tex, ws);
  //calcPixelsGlInterop(tex, ws);

  while (!workshop.shouldStop())
  {
    workshop.beginFrame();

    ImGui::Begin("Main");
    static bool shouldShowImGuiDemo = false;
    ImGui::Checkbox("Show Demo", &shouldShowImGuiDemo);
    if (shouldShowImGuiDemo)
      ImGui::ShowDemoWindow();

    static glm::vec3 bgColor{42 / 256.0, 96 / 256.0, 87 / 256.0};
    ImGui::ColorEdit3("BG Color", glm::value_ptr(bgColor));
    ImGui::End();

    // VAO binding is needed in 4.6 was not needed in 3.1
    uint32_t vao;
    glGenVertexArrays(1, &vao);
    glBindVertexArray(vao);

    glClearColor(bgColor.x, bgColor.y, bgColor.z, 1);
    glClear(GL_COLOR_BUFFER_BIT);

    const auto winSize = workshop.getWindowSize();
    glViewport(0, 0, winSize.x, winSize.y);

    shader.bind();
    tex.bind();
    glDrawArrays(GL_TRIANGLES, 0, 6);
    tex.unbind();
    shader.unbind();

    workshop.endFrame();
  }

  return 0;
}