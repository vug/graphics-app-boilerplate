#include "hip/hip_runtime.h"
#include "kernels.h"

#include <hip/hip_complex.h>

#include <stdio.h>

__global__ void vectorAdd(float* a, float* b, float* out) {
  size_t ix = threadIdx.x;
  out[ix] = a[ix] + b[ix];
}

__global__ void genTexture(unsigned int* pixels, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int idx = y * width + x;

  if (x >= width || y >= height)
    return;

  unsigned char red = x * 255 / width;
  unsigned char green = y * 255 / height;
  unsigned char blue = 0;
  unsigned char alpha = 255;
  pixels[idx] = (alpha << 24) + (blue << 16) + (green << 8) + red;

  if (idx >= 480000) {
    printf("r: (%d, %d), idx: %d, block: (%d, %d), thread: (%d %d), %X\n", x, y, idx, blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, pixels[idx]);
  }
}

void launchGenTexture(unsigned int* pixels, int width, int height) {
  const auto threadSize = dim3(32, 32);
  const auto blockSize = dim3(width / threadSize.x + 1, height / threadSize.y + 1);
  printf("numPixels: %d, sizeBytes: %zd, blockSize (%d, %d), threadSize: (%d, %d)\n", width * height, width * height * sizeof(unsigned int), blockSize.x, blockSize.y, threadSize.x, threadSize.y);
  genTexture<<<blockSize, threadSize>>>(pixels, width, height);
}

__global__ void genSurface(hipSurfaceObject_t surf, int width, int height, int timeStep) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  if (x >= width || y >= height)
    return;

  unsigned char red = ((x + timeStep) % width) * 255 / width;
  unsigned char green = y * 255 / height;
  unsigned char blue = 0;
  unsigned char alpha = 255;
  uchar4 pixel{red, green, blue, alpha};
  surf2Dwrite(pixel, surf, x * sizeof(uchar4), y);  // TODO: learn why x * 4 but just y
}

void launchGenSurface(hipSurfaceObject_t surf, int width, int height, int timeStep) {
  const auto threadSize = dim3(32, 32);
  const auto blockSize = dim3(width / threadSize.x + 1, height / threadSize.y + 1);
  genSurface<<<blockSize, threadSize>>>(surf, width, height, timeStep);
}

__global__ void genMandelbrot(hipSurfaceObject_t surf, int texWidth, int texHeight, Model model, int maxIter, bool useDouble, int timeStep) {
  int texX = blockIdx.x * blockDim.x + threadIdx.x;
  int texY = blockIdx.y * blockDim.y + threadIdx.y;
  if (texX >= texWidth || texY >= texHeight)
    return;

  double u = (double)texX / (double)texWidth - 0.5; // [0, 1)
  double v = (double)texY / (double)texHeight - 0.5; // [0, 1)
  double width = model.height / texHeight * texWidth;
  double x = model.topLeft.x + u * width;
  double y = model.topLeft.y + v * model.height;
  bool bounded = true;
  int nSteps = 0;
  if (useDouble) {
    hipDoubleComplex z0 = make_hipDoubleComplex(model.z0.x, model.z0.y);
    hipDoubleComplex coord = make_hipDoubleComplex(x, y);
    hipDoubleComplex z = model.fractalType == Fractal_Mandelbrot ? z0 : coord;
    for (int i = 0; i < maxIter; i++) {
      z = model.fractalType == Fractal_Mandelbrot ? hipCadd(hipCmul(z, z), coord) : hipCadd(hipCmul(z, z), z0);
      ++nSteps;
      if (hipCabs(z) > 2.) {
        bounded = false;
        break;
      }
    }
  } else {
    hipFloatComplex z0 = make_hipFloatComplex(model.z0.x, model.z0.y);
    hipFloatComplex coord = make_hipFloatComplex(x, y);
    hipFloatComplex z = model.fractalType == Fractal_Mandelbrot ? z0 : coord;
    for (int i = 0; i < maxIter; i++) {
      z = model.fractalType == Fractal_Mandelbrot ? hipCaddf(hipCmulf(z, z), coord) : hipCaddf(hipCmulf(z, z), z0);
      ++nSteps;
      if (hipCabsf(z) > 2.) {
        bounded = false;
        break;
      }
    }
  }

  unsigned char val = bounded ? 0 : 255 * (maxIter - nSteps) / maxIter;
  unsigned char red = val;
  unsigned char green = val;
  unsigned char blue = val;
  unsigned char alpha = val;
  uchar4 pixel{red, green, blue, alpha};
  surf2Dwrite(pixel, surf, texX * sizeof(uchar4), texY);
}

void launchGenMandelbrot(hipSurfaceObject_t surf, int texWidth, int texHeight, Model model, int maxIter, bool useDouble, int timeStep) {
  const auto threadSize = dim3(32, 32);
  const auto blockSize = dim3(texWidth / threadSize.x + 1, texHeight / threadSize.y + 1);
  genMandelbrot<<<blockSize, threadSize>>>(surf, texWidth, texHeight, model, maxIter, useDouble, timeStep);
}