#include "hip/hip_runtime.h"
#include "kernels.h"

#include <hip/hip_complex.h>

#include <stdio.h>

__global__ void genMandelbrot(hipSurfaceObject_t surf, int texWidth, int texHeight, Model model, int maxIter, bool useDouble, int timeStep) {
  int texX = blockIdx.x * blockDim.x + threadIdx.x;
  int texY = blockIdx.y * blockDim.y + threadIdx.y;
  if (texX >= texWidth || texY >= texHeight)
    return;

  double u = (double)texX / (double)texWidth - 0.5; // [0, 1)
  double v = (double)texY / (double)texHeight - 0.5; // [0, 1)
  double width = model.height / texHeight * texWidth;
  double x = model.topLeft.x + u * width;
  double y = model.topLeft.y + v * model.height;
  bool bounded = true;
  int nSteps = 0;
  if (useDouble) {
    hipDoubleComplex z0 = make_hipDoubleComplex(model.z0.x, model.z0.y);
    hipDoubleComplex coord = make_hipDoubleComplex(x, y);
    hipDoubleComplex z = model.fractalType == Fractal_Mandelbrot ? z0 : coord;
    for (int i = 0; i < maxIter; i++) {
      z = model.fractalType == Fractal_Mandelbrot ? hipCadd(hipCmul(z, z), coord) : hipCadd(hipCmul(z, z), z0);
      ++nSteps;
      if (hipCabs(z) > 2.) {
        bounded = false;
        break;
      }
    }
  } else {
    hipFloatComplex z0 = make_hipFloatComplex(model.z0.x, model.z0.y);
    hipFloatComplex coord = make_hipFloatComplex(x, y);
    hipFloatComplex z = model.fractalType == Fractal_Mandelbrot ? z0 : coord;
    for (int i = 0; i < maxIter; i++) {
      z = model.fractalType == Fractal_Mandelbrot ? hipCaddf(hipCmulf(z, z), coord) : hipCaddf(hipCmulf(z, z), z0);
      ++nSteps;
      if (hipCabsf(z) > 2.) {
        bounded = false;
        break;
      }
    }
  }

  unsigned char val = bounded ? 0 : 255 * (maxIter - nSteps) / maxIter;
  unsigned char red = val;
  unsigned char green = val;
  unsigned char blue = val;
  unsigned char alpha = val;
  uchar4 pixel{red, green, blue, alpha};
  surf2Dwrite(pixel, surf, texX * sizeof(uchar4), texY);
}

void launchGenMandelbrot(hipSurfaceObject_t surf, int texWidth, int texHeight, Model model, int maxIter, bool useDouble, int timeStep) {
  const auto threadSize = dim3(32, 32);
  const auto blockSize = dim3(texWidth / threadSize.x + 1, texHeight / threadSize.y + 1);
  genMandelbrot<<<blockSize, threadSize>>>(surf, texWidth, texHeight, model, maxIter, useDouble, timeStep);
}